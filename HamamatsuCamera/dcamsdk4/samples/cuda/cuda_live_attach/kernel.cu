#include "hip/hip_runtime.h"
// include CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define USE_ZEROCOPYMEMORY	0	// 0: use normal memory reserved by hipHostAlloc(Default), 1: use zero copy memory reserved by hipHostAlloc(Mapped).

__global__	void
blocksum( WORD *src, DWORD* dst, unsigned int n, DWORD *result )
{
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx >= n )	return;

	dst[ idx ] = src[ idx ];		// first 
	__syncthreads();

	for( unsigned int stride = 1; stride < blockDim.x; stride *= 2 )
	{
		if( tid % ( 2 * stride ) == 0 )
		{
			dst[ idx ] += dst[ idx + stride ];
		}
		__syncthreads();
	}

	if( tid == 0 )
	{
		result[ blockIdx.x ] = dst[ blockIdx.x * blockDim.x ];
	}
}

// calculate average center point by GPU
extern "C"
double calc_average_gpu( const void* buf, long rowbytes, long width, long height )
{
	WORD* g_src;
#if USE_ZEROCOPYMEMORY
	checkCudaErrors( hipHostGetDevicePointer( (void**)&g_src, (void*)buf, 0 ) );		// get mapped pointer
#else
	long	framebytes = rowbytes * height;
	checkCudaErrors( hipMalloc( (void**)&g_src, framebytes ) );						// allocate GPU memory
	checkCudaErrors( hipMemcpy( g_src, buf, framebytes, hipMemcpyHostToDevice ) );	// copy CPU memory to GPU
#endif

	// breakdown into threads and blocks.
	DWORD	threadNum	= 256;
	DWORD	blockNum;

	DWORD	matrixNum = width * height;
	if( matrixNum % threadNum )
	{
		matrixNum += ( threadNum - matrixNum % threadNum );
	}

	blockNum = matrixNum / threadNum;

	DWORD*	g_dst;
	checkCudaErrors( hipMalloc( (void**)&g_dst, sizeof(DWORD) * matrixNum ) );
	checkCudaErrors( hipMemset( g_dst, 0, sizeof(DWORD) * matrixNum ) );

	DWORD*	g_blockSum;
	checkCudaErrors( hipMalloc( (void**)&g_blockSum, sizeof(DWORD) * blockNum ) );

	// Run Kernel
	dim3	threads( threadNum );									// thread number
	dim3	grid( blockNum );										// block number

	blocksum<<< grid, threads >>>( g_src, g_dst, (unsigned int)matrixNum, g_blockSum );		// execute in all threads

	// Copy result from GPU to CPU
	DWORD* h_blockSum = (DWORD*)malloc( sizeof(DWORD) * blockNum );
	checkCudaErrors( hipMemcpy( h_blockSum, g_blockSum, sizeof(DWORD) * blockNum, hipMemcpyDeviceToHost ) );

	double	total = 0;

	int	j;
	for( j=0; j<(long)blockNum; j++ )
	{
		total += h_blockSum[j];
	}

	checkCudaErrors( hipFree( g_dst		) );
	checkCudaErrors( hipFree( g_blockSum	) );

#if USE_ZEROCOPYMEMORY
	// nothing to do
#else
	checkCudaErrors( hipFree( g_src		) );					// release GPU memory
#endif

	return total / width / height;
}

extern "C"
BOOL allocBuffer( void** buf, long bufsize )
{
#if USE_ZEROCOPYMEMORY
	// Setup Device
	long dev = 0;
	checkCudaErrors( hipSetDevice( dev ) );

	// Get Device Property
	hipDeviceProp_t	deviceProp;
	checkCudaErrors( hipGetDeviceProperties( &deviceProp, dev ) );

	// Check Zero Copy Memory Supported
	if( ! deviceProp.canMapHostMemory )
	{
		printf( "Device %d does not support mapping CPU host memory.\n", dev );
		checkCudaErrors( hipDeviceReset() );
		return FALSE;
	}

	checkCudaErrors( hipHostAlloc( buf, bufsize, hipHostMallocMapped ) );			// allocate CPU memory with mapping
#else
	checkCudaErrors( hipHostAlloc( buf, bufsize, hipHostMallocDefault ) );			// allocate CPU memory without mapping
#endif
	memset( *buf, 0, bufsize );

	return TRUE;
}

extern "C"
void releaseBuffer( void* buf )
{
	checkCudaErrors( hipHostFree( buf ) );											// release host memory
}